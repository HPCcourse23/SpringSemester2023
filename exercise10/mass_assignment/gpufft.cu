#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include "gpufft.h"

static const char *cufftGetErrorString(hipfftResult cufft_error_type)
{

    switch (cufft_error_type)
    {

    case HIPFFT_SUCCESS:
        return "HIPFFT_SUCCESS: The CUFFT operation was performed";

    case HIPFFT_INVALID_PLAN:
        return "HIPFFT_INVALID_PLAN: The CUFFT plan to execute is invalid";

    case HIPFFT_ALLOC_FAILED:
        return "HIPFFT_ALLOC_FAILED: The allocation of data for CUFFT in memory failed";

    case HIPFFT_INVALID_TYPE:
        return "HIPFFT_INVALID_TYPE: The data type used by CUFFT is invalid";

    case HIPFFT_INVALID_VALUE:
        return "HIPFFT_INVALID_VALUE: The data value used by CUFFT is invalid";

    case HIPFFT_INTERNAL_ERROR:
        return "HIPFFT_INTERNAL_ERROR: An internal error occurred in CUFFT";

    case HIPFFT_EXEC_FAILED:
        return "HIPFFT_EXEC_FAILED: The execution of a plan by CUFFT failed";

    case HIPFFT_SETUP_FAILED:
        return "HIPFFT_SETUP_FAILED: The setup of CUFFT failed";

    case HIPFFT_INVALID_SIZE:
        return "HIPFFT_INVALID_SIZE: The size of the data to be used by CUFFT is invalid";

    case HIPFFT_UNALIGNED_DATA:
        return "HIPFFT_UNALIGNED_DATA: The data to be used by CUFFT is unaligned in memory";
    }

    return "Unknown CUFFT Error";
}

void compute_fft_2D_R2C(blitz::Array<float, 2> &grid, void *data)
{
    int n[] = {grid.rows(), grid.cols()}; // 2D FFT of length NxN
    int inembed[] = {grid.rows(), 2 * (grid.cols() / 2 + 1)};
    int onembed[] = {grid.rows(), (grid.cols() / 2 + 1)};
    int howmany = 1;
    int odist = grid.rows() * (grid.cols() / 2 + 1); // Output distance is in "complex"
    int idist = 2 * odist;                           // Input distance is in "real"
    int istride = 1;                                 // Elements of each FFT are adjacent
    int ostride = 1;

    hipfftHandle plan;
    hipfftPlanMany(&plan, sizeof(n) / sizeof(n[0]), n,
                  inembed, istride, idist,
                  onembed, ostride, odist,
                  HIPFFT_R2C, howmany);

    //    hipfftComplex *data;
    auto data_size = sizeof(hipfftComplex) * howmany * grid.rows() * (grid.cols() / 2 + 1);
    //    hipMalloc((void**)&data, data_size);
    hipMemcpy(data, grid.dataFirst(), data_size, hipMemcpyHostToDevice);

    auto status = hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(data), reinterpret_cast<hipfftComplex *>(data));
    if (status != HIPFFT_SUCCESS)
    {
        const char *errorString = cufftGetErrorString(status);
        printf("CUDA cuFFT Error: %s\n", errorString);
    }

    hipMemcpy(grid.dataFirst(), data, data_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    //    hipFree(data);
    hipfftDestroy(plan);
}

void *allocate_cuda_slab(size_t nGrid)
{
    void *cuda_slab;
    auto slab_size = sizeof(hipfftComplex) * nGrid * (nGrid / 2 + 1);
    hipMalloc((void **)&cuda_slab, slab_size);
    return cuda_slab;
}
