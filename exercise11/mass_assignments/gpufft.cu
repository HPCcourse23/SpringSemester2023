#include "gpufft.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

static const char *cufftGetErrorString(hipfftResult cufft_error_type)
{

    switch (cufft_error_type)
    {

    case HIPFFT_SUCCESS:
        return "HIPFFT_SUCCESS: The CUFFT operation was performed";

    case HIPFFT_INVALID_PLAN:
        return "HIPFFT_INVALID_PLAN: The CUFFT plan to execute is invalid";

    case HIPFFT_ALLOC_FAILED:
        return "HIPFFT_ALLOC_FAILED: The allocation of data for CUFFT in memory failed";

    case HIPFFT_INVALID_TYPE:
        return "HIPFFT_INVALID_TYPE: The data type used by CUFFT is invalid";

    case HIPFFT_INVALID_VALUE:
        return "HIPFFT_INVALID_VALUE: The data value used by CUFFT is invalid";

    case HIPFFT_INTERNAL_ERROR:
        return "HIPFFT_INTERNAL_ERROR: An internal error occurred in CUFFT";

    case HIPFFT_EXEC_FAILED:
        return "HIPFFT_EXEC_FAILED: The execution of a plan by CUFFT failed";

    case HIPFFT_SETUP_FAILED:
        return "HIPFFT_SETUP_FAILED: The setup of CUFFT failed";

    case HIPFFT_INVALID_SIZE:
        return "HIPFFT_INVALID_SIZE: The size of the data to be used by CUFFT is invalid";

    case HIPFFT_UNALIGNED_DATA:
        return "HIPFFT_UNALIGNED_DATA: The data to be used by CUFFT is unaligned in memory";
    }

    return "Unknown CUFFT Error";
}

// Create a plan to do a 2D transform for the given grid (in-place)
hipfftHandle make_plan(int nGrid, int n_streams, stream_info info[])
{
    int n[] = {nGrid, nGrid}; // 2D FFT of length NxN
    int inembed[] = {nGrid, 2 * (nGrid / 2 + 1)};
    int onembed[] = {nGrid, nGrid / 2 + 1};
    int howmany = 1;
    int odist = onembed[0] * onembed[1]; // Output distance is in "complex"
    int idist = 2 * odist;               // Input distance is in "real"
    int istride = 1;                     // Elements of each FFT are adjacent
    int ostride = 1;
    size_t workSize;

    hipfftHandle plan;
    hipfftCreate(&plan);
    hipfftSetAutoAllocation(plan, 0);
    hipfftMakePlanMany(plan, sizeof(n) / sizeof(n[0]), n,
                      inembed, istride, idist,
                      onembed, ostride, odist,
                      HIPFFT_R2C, howmany, &workSize);
    for (auto i = 0; i < n_streams; ++i)
    {
        auto slab_size = sizeof(hipfftComplex) * onembed[0] * onembed[1];
        hipStreamCreate(&info[i].stream);
        hipMallocAsync(&info[i].slab, slab_size, info[i].stream);
        hipMallocAsync(&info[i].work, workSize, info[i].stream);
    }

    return plan;
}

void compute_fft_2D_R2C(blitz::Array<float, 2> &grid, stream_info *info, hipfftHandle plan)
{

    auto data_size = sizeof(hipfftComplex) * grid.rows() * (grid.cols() / 2 + 1);
    hipMemcpyAsync(info->slab, grid.dataFirst(), data_size, hipMemcpyHostToDevice, info->stream);
    hipfftSetStream(plan, info->stream);
    hipfftSetWorkArea(plan, info->work);
    auto status = hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(info->slab), reinterpret_cast<hipfftComplex *>(info->slab));
    if (status != HIPFFT_SUCCESS)
    {
        const char *errorString = cufftGetErrorString(status);
        printf("CUDA cuFFT Error: %s\n", errorString);
    }
    hipMemcpyAsync(grid.dataFirst(), info->slab, data_size, hipMemcpyDeviceToHost, info->stream);
}

